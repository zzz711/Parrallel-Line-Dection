#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>

__global__ void  whiteDect(const uchar4*  rgbaImage, unsigned char* const newImage,  int numRows, int numCols){
	int col = threadIdx.x;
	int row = blockIdx.x;
	int xID = col + row * numRows;
	uchar4 linedImage = rgbaImage[xID];	

	if(rgbaImage[xID].x < 100 && rgbaImage[xID].y < 100 && rgbaImage[xID].z < 100){//if  pixel isn't fully white
		newImage[xID] = linedImage.x * 0 + linedImage.y * 0 + linedImage.z*0; //make pixel black
	}
	
	__syncthreads();
}

__global__ void whiteToRed(const uchar4*  rgbaImage, unsigned char* const newImage,  int numRows, int numCols){
	int col = threadIdx.x;
	int row = blockIdx.x;
	int xID = col + row * numRows;
	uchar4 linedImage = rgbaImage[xID];	

	if( rgbaImage[xID].x == 255 && rgbaImage[xID].y == 255 && rgbaImage[xID].z == 255){
		newImage[xID] = linedImage.x * 255 + linedImage.y * 100 + linedImage.z * 50; //white to red
	}

	__syncthreads();
}

void lineDetect(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage, unsigned char* const d_greyImage, size_t numRows, size_t numCols) {
	const dim3 blockSize(numRows, 1, 1);
	const dim3 gridSize( numCols, 1, 1); 

	whiteDect<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
	 hipDeviceSynchronize();
	whiteToRed<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
	 hipDeviceSynchronize();
	 checkCudaErrors(hipGetLastError());
}

