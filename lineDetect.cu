#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>

__global__ void  whiteDect(const uchar4* const rgbaImage, unsigned char* const newImage,  int numRows, int numCols){
	int col = threadIdx.x;
	int row = blockIdx.x;
	int xID = col + row * numRows;
	uchar4 linedImage = rgbaImage[xID];	

	if(rgbaImage[xID].x != 255 && rgbaImage[xID].y != 255 && rgbaImage[xID].z != 255){//if  pixel isn't fully white
		newImage[xID] = linedImage.x * 0 + linedImage.y * 0 + linedImage.z*0; //make pixel black
	}

}

__global__ void whiteToRed(const uchar4* const rgbaImage, unsigned char* const newImage,  int numRows, int numCols){
	int col = threadIdx.x;
	int row = blockIdx.x;
	int xID = col + row * numRows;
	uchar4 linedImage = rgbaImage[xID];	

	if( rgaImage[xID].x == 255 && rgaImage[xID].y == 255 && rgaImage[xID].z == 255){
		newImage[xID] = linedImage.x * 1 + linedImage.y * 0 + linedImage.z * 0; //white to red
	}

}

void lineDetect(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage, unsigned char* const d_greyImage, size_t numRows, size_t numCols) {
	const dim3 blockSize(numRows, 1, 1);
	const dim3 gridSize( numCols, 1, 1); 

	whiteDect<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
	__syncthreads();
	whiteToRed<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
	__syncthreads();
	 hipDeviceSynchronize();
	 checkCudaErrors(hipGetLastError());
}

