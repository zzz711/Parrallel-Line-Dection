#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>

__global__  whiteDect(const uchar4* const rgbaImage, unsigned char* const newImage,  int numRows, int numCols){
	int threadsPerBlock = blockDim.x * blockDim.y;
	int blockId = blockIdx.y + (blockIdx.x * gridDim.y);
	int threadId = threadIdx.y + (threadIdx.x * blockDim.y);

	int offset = (blockId * threadsPerBlock) + threadId;;

	if(rgbaImage[offset].x == 255 && rgbaImage[offset].y == 255 && rgbaImage[offset].z == 255){//if  pixel is fully white. what if they're not fully white?
		//do something
	}
	else{
		newImag = 


