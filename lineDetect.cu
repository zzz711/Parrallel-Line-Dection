#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>

__global__  whiteDect(const uchar4* const rgbaImage, unsigned char* const newImage,  int numRows, int numCols){
	int col = threadIdx.x;
	int row = blockIdx.x;
	int xID = col + row * numRows;
	uchar4 linedImage = rgbaImage[xID];	

	if(rgbaImage[xID].x != 255 && rgbaImage[xID].y != 255 && rgbaImage[xID].z != 255){//if  pixel isn't fully white
		newImage[xID] = linedImage.x * 0 + linedImage.y * 0 + linedImage.z*0; //make pixel black
	}

}

__global__  whiteToRed(const uchar4* const rgbaImage, unsigned char* const newImage,  int numRows, int numCols){
	int col = threadIdx.x;
	int row = blockIdx.x;
	int xID = col + row * numRows;
	uchar4 linedImage = rgbaImage[xID];	

	if( rgaImage[xID].x == 255 && rgaImage[xID].y == 255 && rgaImage[xID].z == 255){
		newImage[xID] = linedImage.x * 1 + linedImage.y * 0 + linedImage.z * 0; //white to red
	}

}

void lineDetect(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage, unsigned char* const d_greyImage, size_t numRows, size_t numCols) {


